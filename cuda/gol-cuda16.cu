
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define BUFSIZE 64
#define BLOCK_SIZE 16

// Perdiodicty Preservation retains our periodicity
// Runs on CPU 
void periodicityPreservationCPU(int N, char *cells)
{
    int i;
    //rows
    for (i = 1; i <= N; ++i)
    {
        //Copy first real row to bottom extra row
        cells[(N+2)*(N+1)+i] = cells[(N+2)+i];
        //Copy last real row to top extra row
        cells[i] = cells[(N+2)*N + i];
    }
    //cols
    for (i = 0; i <= N+1; ++i)
    {
        //Copy first real column to right last extra column
        cells[i*(N+2)+N+1] = cells[i*(N+2)+1];
        //Copy last real column to left last extra column 
        cells[i*(N+2)] = cells[i*(N+2) + N];  
    }
}

// Runs on GPU
__global__ void periodicityPreservationGPU(int N, char *cells)
{
    int i;
    //rows
    for (i = 1; i <= N; ++i)
    {
        //Copy first real row to bottom extra row
        cells[(N+2)*(N+1)+i] = cells[(N+2)+i];
        //Copy last real row to top extra row
        cells[i] = cells[(N+2)*N + i];
    }
    //cols
    for (i = 0; i <= N+1; ++i)
    {
        //Copy first real column to right last extra column
        cells[i*(N+2)+N+1] = cells[i*(N+2)+1];
        //Copy last real column to left last extra column 
        cells[i*(N+2)] = cells[i*(N+2) + N];  
    }
}


/* Our evolve kernels shoulder our evolutions procedure ,
   specifically [GridSize] blocks X [256] threads run towards 
   our evolve kernels . Each thread looks after the evolution
   of one cell by calculating each neighbors and abiding 
   the evolution rules */

// Based on global memoery
__global__ void evovle_kernel(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
    // Achieve indexng on 2D blocks
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    // Thread calculates its global id
    int id = ix * (N+2) + iy;
 
    int neighbors;
 
    if (ix <= N && iy <= N) {
        neighbors = oldGen[id+(N+2)] + oldGen[id-(N+2)]     //lower upper
                    + oldGen[id+1] + oldGen[id-1]           //right left
                    + oldGen[id+(N+3)] + oldGen[id-(N+3)]   //diagonals
                    + oldGen[id-(N+1)] + oldGen[id+(N+1)];
 
        char cell  = oldGen[id];
        newGen[id] = neighbors == 3 || (neighbors == 2 && cell); // Fill in the cells

        // Terminating Checkings
        if (newGen[id] != 0) (*allzeros)++;             // Check if all cells are dead
        if (newGen[id] != oldGen[id]) (*change)++;      // Check if life stayed the same

    } 
}


// With the help of shared memory
__global__ void evovle_kernel_shared(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
	// Global
    int ix = (blockDim.x - 2) * blockIdx.x + threadIdx.x;       //Different indexing as we declared more blocks (see SideGrid)
    int iy = (blockDim.y - 2) * blockIdx.y + threadIdx.y;
    int id = ix * (N+2) + iy;
 
    int i = threadIdx.x;
    int j = threadIdx.y;
    int neighbors;
 
    // Declare the shared memory on a per block level
    __shared__ char oldGen_shared[BLOCK_SIZE][BLOCK_SIZE];
 
    // Copy cells into shared memory
    if (ix <= N+1 && iy <= N+1)
        oldGen_shared[i][j] = oldGen[id];           //Copy each cell and in the sides of shared array the blocks' neighbors
 
    // Sync threads on block
    __syncthreads();
 
    if (ix <= N && iy <= N) {
        if(i != 0 && i != (blockDim.y-1) && j != 0 && j != (blockDim.x-1)) {

            // Get the number of neighbors for a given oldGen point
            neighbors = oldGen_shared[i+1][j] + oldGen_shared[i-1][j]         //lower upper
                    + oldGen_shared[i][j+1] + oldGen_shared[i][j-1]           //right left
                    + oldGen_shared[i+1][j+1] + oldGen_shared[i-1][j-1]       //diagonals
                    + oldGen_shared[i-1][j+1] + oldGen_shared[i+1][j-1];
 
            char cell  = oldGen_shared[i][j];
			newGen[id] = neighbors == 3 || (neighbors == 2 && cell); // Fill in  the cells

            // Terminating Checkings
            if (newGen[id] != 0) (*allzeros)++;        // Check if all cells are dead
            if (newGen[id] != oldGen[id]) (*change)++; // Check if life stayed the same
        }
    }
}



int main(int argc, char* argv[])
{    
    int i, j;
    int N;              // Dimension of cells 
    int generations;    // Generations of evolution
    FILE *fp = NULL;    // A file for input (optional)
    int shared = 0;     // Use share memory or not  
    int output = 0;     // Print the array in every generation, at the end or not at all
    int periodicity = 1;   // Choose if we want the calculate the periodicity of side cells in cpu or gpu
    int doom = 0 ; 		// With terminal checking or Not 

    /*Read the arguments*/
    for (i = 0; i < argc; i++){
        if (!strcmp(argv[i], "-n")) N = atoi(argv[++i]);
        else if (!strcmp(argv[i], "-g")) generations = atoi(argv[++i]);
        else if (!strcmp(argv[i], "-i")) fp = fopen(argv[++i], "r");
        else if (!strcmp(argv[i], "-s")) shared = atoi(argv[++i]);
        else if (!strcmp(argv[i], "-p")) periodicity = atoi(argv[++i]);
        else if (!strcmp(argv[i], "-o1")) output = 1;
        else if (!strcmp(argv[i], "-o2")) output = 2;
        else if (!strcmp(argv[i], "-d")) doom = atoi(argv[++i]);
    }

    // Definitions of one dimension arrays on host and device
    // Actually are a 2D size array but we declare them 1D as we want contiguous memory allocation
    char* h_cells; // our results will be copied on CPU cells
    char* d_old;   // Device 2D cells for oldGen cells
    char* d_new;   // Device 2D cells for new generations cells
    char* d_Swap;  // Swap cells just like game_mpi
    
    // Allocation of host cells, we allocate more byte space [(N+2)^2], to retain our periodicity of cells
    int bytes = sizeof(char)*(N+2)*(N+2);
    h_cells   = (char*)malloc(bytes);
 
    // If we don't have a file, fill it with mighty randomness
    if (fp == NULL)
    {
        srand(time(NULL));
        for(i = 1; i<=N; i++) {
            for(j = 1; j<=N; j++) {
                h_cells[i*(N+2)+j] = rand() % 2;
            }
        }
    }
    else // fill the cells from file
    {
        /*Read from input file the position of initial live cells (if there is input file)*/
        if (fp != NULL){
            char line[BUFSIZE], *token, delim[2] = " ";
            fgets(line, BUFSIZE, fp);
            while (!feof(fp)){                  //Till the end of the file read from it
                token = strtok(line, delim);
                i = atoi(token);
                token = strtok(NULL, delim);
                j = atoi(token);
                h_cells[i*(N+2)+j] = 1;         //Assign a live cell in the coordinates given
                fgets(line, BUFSIZE, fp);
            }
            fclose(fp); //Close the file
        }
    }

    // Start Timer After Initialising The Array
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
	hipEventCreate(&event2);

	// Start
	hipEventRecord(event1, 0); //where 0 is the default stream

    // Allocate device arrays on GPU memory
    hipMalloc(&d_old, bytes);
    hipMalloc(&d_new, bytes);

    int *h_allzeros, *h_change;
    int *d_allzeros, *d_change;
    h_allzeros = (int*)malloc(sizeof(int));
    h_change   = (int*)malloc(sizeof(int));
    hipMalloc(&d_allzeros, sizeof(int));
    hipMalloc(&d_change, sizeof(int));

    // Let's fill our device cells 
    hipMemcpy(d_old, h_cells, bytes, hipMemcpyHostToDevice);
 
    // Set the prefferes cache configuration for the device function if we want to use shared memory
    if (shared == 1) hipFuncSetCacheConfig(reinterpret_cast<const void*>(evovle_kernel_shared), hipFuncCachePreferShared);

    // Find the Blocks each side of Grid has (e.g For N = 128 , we will need 8 blocks on each side of the grid)
    int SideGrid;
    if (shared)
        SideGrid = (int)ceil(N/(float)(BLOCK_SIZE-2));       //For easier copy in shared memory, we declare more blocks per side of Grid
    else
        SideGrid = (int)ceil(N/(float)BLOCK_SIZE);

 	// For the evolution kernel we specify a two dimensional block size , 16x16 size , 256 threads 
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
    
    // Create a 2D Grid , to hold our blocks
    // e.g. N = 128 , we will call four 8x8 grid = 64 blocks , 64 x 256 = 16384 threads for the N x N = 16384 cells
    dim3 gridSize(SideGrid, SideGrid, 1);


    // Start Evolutioning for given generations
    for (i = 0; i < generations; i++)
    {
        // Print the state of our cells (if told so in command line flags)
        if (output == 2 || (output == 1 && i == (generations-1)))
        {
            hipMemcpy(h_cells, d_old, bytes, hipMemcpyDeviceToHost);
            int r, c;
            printf("\n///////////////////////////////////////////////////\n\n");
            for (r = 1; r <= N; r++){
                for (c = 1; c <= N; c++){
                    if (h_cells[r*(N+2)+c] == 0) printf("-");
                    else if (h_cells[r*(N+2)+c] == 1) printf("X");
                    else printf("?");
                }
                printf("\n");
            }
        }

        if (i != (generations-1))
        {
            if ((doom == 1) && ((i%10) == 0) && (i != 0)){
                *h_allzeros = 0;
                *h_change   = 0;
                hipMemcpy(d_allzeros, h_allzeros, sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(d_change, h_change, sizeof(int), hipMemcpyHostToDevice);
            }

            if (periodicity == 0){                                       // Calculate periodicity on cpu (faster processor but expensive memcopy)
                hipMemcpy(h_cells, d_old, bytes, hipMemcpyDeviceToHost);
                periodicityPreservationCPU(N, h_cells);                      // CPU is better if N is small (due to for loop) (CPU-GHz GPU-MHz)
                hipMemcpy(d_old, h_cells, bytes, hipMemcpyHostToDevice);   // GPU is better in big numbers as we avoid transfer data (memcopy)
            }
            else periodicityPreservationGPU<<<1,1>>>(N, d_old);    // else on gpu (no memcopy on cpu but much slower gpu processor)

            // Evolution of the cells, using shared memory in gpu or not
            if (shared)
                evovle_kernel_shared<<<gridSize, blockSize>>>(N, d_old, d_new, d_allzeros, d_change);
            else
                evovle_kernel<<<gridSize, blockSize>>>(N, d_old, d_new, d_allzeros, d_change);

            if ((doom == 1) && ((i%10) == 0) && (i != 0)){
                hipMemcpy(h_allzeros, d_allzeros, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(h_change, d_change, sizeof(int), hipMemcpyDeviceToHost);
                //printf("Zeros:%d Change:%d\n", (*h_allzeros), (*h_change));
                if ((*h_change) == 0 || (*h_allzeros) == 0){
                    printf("Program terminated (nothing changed or all extinguisted in this generation)\n");
                    break;
                }
            }

            // Swap our grids and proceed to next generation
            d_Swap = d_old;
            d_old  = d_new;
            d_new  = d_Swap;
        }
    }

    // Copy back results and sum
    hipMemcpy(h_cells, d_old, bytes, hipMemcpyDeviceToHost);
 
    // Release memory
    hipFree(d_allzeros);
    hipFree(d_change);
    hipFree(d_old);
    hipFree(d_new);
    free(h_cells);
    free(h_change);
    free(h_allzeros);

    // Stop The Timer
    hipEventRecord(event2, 0);
 
 	// Calculate Elapsed Time
    //synchronize
	hipEventSynchronize(event1); //optional
	hipEventSynchronize(event2); //wait for the event to be executed!

	//calculate time
	float dt_ms;
	hipEventElapsedTime(&dt_ms, event1, event2);

    printf("--------------------------------------------------------------\n");
    printf("Runtime %f \n", dt_ms/1000);
    printf("--------------------------------------------------------------\n");

    return 0;
}